#include "hip/hip_runtime.h"
/**
 * Universidade de Sao Paulo
 * Instituto de Ciencias Matematicas e de Computacao
 * 
 * Trabalho 3: Smooth
 * Algoritmo usando CUDA
 * 
 * SSC0143 Programacao Concorrente
 * 
 * Elias Italiano Rodrigues 7987251
 * Rodolfo Megiato de Lima  7987286
 * Vinicius Katata Biondo   6783972
 */
#include <stdlib.h>
#include <stdio.h>
#include <getopt.h>
#include <sys/time.h>
#include "../inc/image.h"
#include "../inc/ppm_p3.h"
#include "../inc/smooth.h"

/**
 * Le uma imagem PPM P3 da entrada padrao (vetorizada)
 */
pixel_t *ppm_p3_vscan(ppm_header_t *ppm)
{
	int i, npixels;
	pixel_t *img;

	// Le o cabecalho
	scanf("%s", ppm->format);
	scanf("%d", &ppm->width);
	scanf("%d", &ppm->height);
	scanf("%d", &ppm->max_value);

	npixels = ppm->width * ppm->height;
	
	// Aloca vetor de pixels para guardar a imagem inteira
	img = (pixel_t *)malloc(sizeof(pixel_t) * npixels);
	if (img == NULL) {
		fprintf(stderr, "ERROR: could not allocate memory for image\n");
		exit(EXIT_FAILURE);
	}

	// Le e carrega a imagem para a memoria
	for (i = 0; i < npixels; i++) {
		scanf("%hhu", &img[i][R]);
		scanf("%hhu", &img[i][G]);
		scanf("%hhu", &img[i][B]);
	}
	
	return img;	
}

/**
 * Imprime uma imagem (vetorizada) PPM P3 na saida padrao
 */
void ppm_p3_vprint(pixel_t *img, ppm_header_t *ppm)
{
	int i, npixels = ppm->width * ppm->height;

	printf("%s\n", ppm->format);
	printf("%d %d\n", ppm->width, ppm->height);
	printf("%d\n", ppm->max_value);

	for (i = 0; i < npixels; i++) {
		printf("%hhu ", img[i][R]);
		printf("%hhu ", img[i][G]);
		printf("%hhu ", img[i][B]);
	}
}

/**
 * Funcao que aplica o algoritmo smooth para remocao de ruidos em uma
 * imagem vetorizada usando CUDA
 */
__global__ 
void cudaSmooth(pixel_t *img_in, pixel_t *img_out, int width, int height)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int a, b, k, l;
	int mean[3];

	// Valida posicao
	if (i >= height || j >= width)
		return;
	
	mean[R] = 0;
	mean[G] = 0;
	mean[B] = 0;

	// Itera na regiao do smooth
	for (a = 0, k = i - BLOCK_OFFSET; a < BLOCK_SIZE; a++, k++) {
		for (b = 0, l = j - BLOCK_OFFSET; b < BLOCK_SIZE; b++, l++) {
			if (k < 0 || l < 0 || k >= height || l >= width) {
				// Quando nao houver pixel valido (problema da borda)
				//	coloca o pixel padrao
				mean[R] += DEFAULT_R;
				mean[G] += DEFAULT_G;
				mean[B] += DEFAULT_B;
			} else {
				mean[R] += (int)img_in[k * width + l][R];
				mean[G] += (int)img_in[k * width + l][G];
				mean[B] += (int)img_in[k * width + l][B];
			}
		}
	}

	// Substitui na imagem resultado
	img_out[i * width + j][R] = (unsigned char)(mean[R] / BLOCK_NPIXELS);
	img_out[i * width + j][G] = (unsigned char)(mean[G] / BLOCK_NPIXELS);
	img_out[i * width + j][B] = (unsigned char)(mean[B] / BLOCK_NPIXELS);
}

/**
 * Verifica retorno de um hipError_t
 */
void cudaFatal(hipError_t error)
{
	if (error != hipSuccess) {
		fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

/**
 * Programa principal
 */
int main(int argc, char **argv)
{
	ppm_header_t ppm; // cabecalho do arquivo PPM P3
	pixel_t *img; // imagem inteira (vetorizada) do host
	pixel_t *img_in, *img_out; // imagem inteira (vetorizada) do device
	int npixels;
	
	struct timeval start, end;

	// Opcoes do programa
	int c;
	int do_output = 1, do_time = 0;
	struct option long_options[] = {
		{"no-output", no_argument,  &do_output, 0 },
		{"time",      no_argument,  &do_time,   1 },
		{0, 0, 0, 0}
	};

	// Tratamento dos argumentos usando optget
	while ((c = getopt_long(argc, argv, ":nt", long_options, NULL)) != -1) {
		switch (c) {
			case 'n':
				do_output = 0;
				break;
			case 't':
				do_time = 1;
				break;
			case 0:
			case '?':
			case ':':
			default:
				break;
		}
	}
	
	// Le imagem da entrada padrao
	img = ppm_p3_vscan(&ppm);
	npixels = ppm.width * ppm.height;

	gettimeofday(&start, NULL);

	// Aloca memoria na device
	cudaFatal(hipMalloc(&img_in,  sizeof(pixel_t) * npixels));
	cudaFatal(hipMalloc(&img_out, sizeof(pixel_t) * npixels));
	
	// Copia imagem para a device
	cudaFatal(hipMemcpy(img_in, img, sizeof(pixel_t) * npixels, hipMemcpyHostToDevice));

	// 32*32 = 1024, o valor maximo de quantidade de threads por bloco :D
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks((ppm.width + threadsPerBlock.x - 1) / threadsPerBlock.x, (ppm.height + threadsPerBlock.y - 1 ) / threadsPerBlock.y);

	cudaSmooth<<<numBlocks, threadsPerBlock>>>(img_in, img_out, ppm.width, ppm.height);
	cudaFatal(hipDeviceSynchronize());

	// Copia resultado de volta para o host
	cudaFatal(hipMemcpy(img, img_out, sizeof(pixel_t) * npixels, hipMemcpyDeviceToHost));

	gettimeofday(&end, NULL);

	// Imprime tempo de processamento
	if (do_time) {
		printf("%.2lf\n", (double)(end.tv_sec - start.tv_sec) + (double)(end.tv_usec - start.tv_usec) / 1000000.0 );
	}

	// Imprime imagem apos aplicado o smooth
	if (do_output) {
		ppm_p3_vprint(img, &ppm);
	}

	// Libera memoria
	free(img);
	hipFree(img_in);
	hipFree(img_out);
	
	return EXIT_SUCCESS;
}
